
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printHelloFromGPU(int device, int totalDevices) {
    // Each thread will print the message from its assigned GPU
    printf("Hello! I'm GPU %d out of %d GPUs in total.\n", device, totalDevices);
}

int main() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    
    if (err != hipSuccess || deviceCount == 0) {
        printf("No CUDA-capable GPU detected or CUDA error.\n");
        return 1;
    }

    for (int device = 0; device < deviceCount; ++device) {
        hipSetDevice(device);

        // Launch the kernel on the current device
        printHelloFromGPU<<<1, 1>>>(device, deviceCount);
        
        // Synchronize to ensure the kernel has finished before moving to the next device
        hipDeviceSynchronize();
    }

    return 0;
}

