#include <stdlib.h>
#include <stdio.h>
// cuda includes
#include "hip/hip_runtime.h"


__global__
void dxpy(int n, double *x, double *y, double *z)
{
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * gridDim.x;

  for (size_t i = index; i < n; i += stride)
    z[i] = x[i] + y[i];
}

extern "C" void dxpy_cfcn(int N, double *x, double *y, double *out){
    
    double *d_x, *d_y, *d_out;   
    
    hipMalloc(&d_x, N*sizeof(double));
    hipMalloc(&d_y, N*sizeof(double));
    hipMalloc(&d_out, N*sizeof(double));
   
    hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(double), hipMemcpyHostToDevice);

   // Perform DAXPY on 1M elements
   dxpy<<<320, 256>>>(N, d_x, d_y, d_out);
   
   hipMemcpy(out, d_out, N*sizeof(double), hipMemcpyDeviceToHost);

   hipFree(d_x);
   hipFree(d_y);
   hipFree(d_out);
}
